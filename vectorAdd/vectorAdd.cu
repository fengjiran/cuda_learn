//
// Created by richard on 9/22/24.
//

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <cstdio>
#include <hip/hip_runtime.h>// For the CUDA runtime routines (prefixed with "cuda_")
#include <iostream>
#include <random>
#include <vector>

/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void vectorAdd(const float* a, const float* b, float* c, int numElems) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < numElems) {
        c[i] = a[i] + b[i] + 0.0f;
    }
}

/**
 * Host test routine
 */
int main() {
    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    int numElements = 50000;
    size_t size = numElements * sizeof(float);
    std::cout << "[Vector addition of " << numElements << " elements]\n";

    std::vector<float> hostVecA(numElements);// Allocate the host input vector A
    std::vector<float> hostVecB(numElements);// Allocate the host input vector B
    std::vector<float> hostVecC(numElements);// Allocate the host output vector C

    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<float> dist(0, 1);

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i) {
        hostVecA[i] = dist(gen);
        hostVecB[i] = dist(gen);
    }

    // Allocate the device input vector A
    float* devPtrA = nullptr;
    err = hipMalloc(reinterpret_cast<void**>(&devPtrA), size);
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate device vector A ("
                  << "error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    // Allocate the device input vector A
    float* devPtrB = nullptr;
    err = hipMalloc(reinterpret_cast<void**>(&devPtrB), size);
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate device vector B ("
                  << "error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    // Allocate the device output vector C
    float* devPtrC = nullptr;
    err = hipMalloc(reinterpret_cast<void**>(&devPtrC), size);
    if (err != hipSuccess) {
        std::cerr << "Failed to allocate device vector C ("
                  << "error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    // Copy the host input vectors A and B in host memory to the device input
    // vectors in device memory
    std::cout << "Copy input data from the host memory to the CUDA device\n";
    err = hipMemcpy(devPtrA, hostVecA.data(), size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Failed to copy vector A from host to device ("
                  << "error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(devPtrB, hostVecB.data(), size, hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        std::cerr << "Failed to copy vector B from host to device ("
                  << "error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    // launch the vector add cuda kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (numElements + threadsPerBlock - 1) / threadsPerBlock;
    std::cout << "CUDA kernel launch with " << blocksPerGrid << " blocks of " << threadsPerBlock << " threads\n";
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(devPtrA, devPtrB, devPtrC, numElements);
    err = hipGetLastError();

    if (err != hipSuccess) {
        std::cerr << "Failed to launch vectorAdd kernel ("
                  << "error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    std::cout << "Copy output data from the CUDA device to the host memory\n";
    err = hipMemcpy(hostVecC.data(), devPtrC, size, hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        std::cerr << "Failed to copy vector C from device to host ("
                  << "error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i) {
        if (std::fabs(hostVecA[i] + hostVecB[i] - hostVecC[i]) > std::numeric_limits<float>::epsilon()) {
            std::cerr << "Result verification failed at element " << i << std::endl;
            exit(EXIT_FAILURE);
        }
    }

    std::cout << "Test PASSED\n";

    // Free device global memory
    err = hipFree(devPtrA);
    if (err != hipSuccess) {
        std::cerr << "Failed to free device vector A ("
                  << "error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    err = hipFree(devPtrB);
    if (err != hipSuccess) {
        std::cerr << "Failed to free device vector B ("
                  << "error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }

    err = hipFree(devPtrC);
    if (err != hipSuccess) {
        std::cerr << "Failed to free device vector C ("
                  << "error code " << hipGetErrorString(err) << ")!\n";
        exit(EXIT_FAILURE);
    }


    std::cout << "Done\n";

    return 0;
}