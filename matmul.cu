
#include <hip/hip_runtime.h>
// Naive matmul
__global__ void MatMul_0(const float* A, const float* B, float* C, int M, int N, int K) {
    uint row = blockIdx.y * blockDim.y + threadIdx.y;
    uint col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < M && col < N) {
        float sum = 0;
        for (int i = 0; i < K; i++) {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = sum;
    }
}

void TestMatMul() {
    //
}